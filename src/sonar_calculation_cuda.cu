#include "hip/hip_runtime.h"
/*
 * Copyright 2020 Naval Postgraduate School
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
*/

#include <nps_uw_multibeam_sonar/sonar_calculation_cuda.cuh>

// #include <math.h>
#include <assert.h>

// For complex numbers
#include <thrust/complex.h>
#include <hip/hip_complex.h>

// For rand() function
#include <unistd.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// For FFT
#include <hipfft/hipfft.h>
#include <hipfft/hipfftw.h>
#include <thrust/device_vector.h>
#include <list>

#include <chrono>

#define BLOCK_SIZE 32

static inline void _safe_cuda_call(hipError_t err, const char *msg,
                                   const char *file_name, const int line_number)
{
  if (err != hipSuccess)
  {
    fprintf(stderr, "%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n",
            msg, file_name, line_number, hipGetErrorString(err));
    std::cin.get();
    exit(EXIT_FAILURE);
  }
}

#define SAFE_CALL(call, msg) _safe_cuda_call((call), (msg), __FILE__, __LINE__)

///////////////////////////////////////////////////////////////////////////
// Incident Angle Calculation Function
// incidence angle is target's normal angle accounting for the ray's azimuth
// and elevation
__device__ float compute_incidence(float azimuth, float elevation, float *normal)
{
  // ray normal from camera azimuth and elevation
  float camera_x = cosf(-azimuth) * cosf(elevation);
  float camera_y = sinf(-azimuth) * cosf(elevation);
  float camera_z = sinf(elevation);
  float ray_normal[3] = {camera_x, camera_y, camera_z};

  // target normal with axes compensated to camera axes
  float target_normal[3] = {normal[2], -normal[0], -normal[1]};

  // dot product
  float dot_product = ray_normal[0] * target_normal[0]
                      + ray_normal[1] * target_normal[1]
                      + ray_normal[2] * target_normal[2];

  return M_PI - acosf(dot_product);
}

///////////////////////////////////////////////////////////////////////////
__device__ __host__ float unnormalized_sinc(float t)
{
  if (abs(t) < 1E-8)
    return 1.0;
  else
    return sin(t) / t;
}

///////////////////////////////////////////////////////////////////////////
template <typename T>
__global__ void column_sums_reduce(const T *__restrict__ in, T *__restrict__ out, size_t width, size_t height)
{

  __shared__ T sdata[BLOCK_SIZE][BLOCK_SIZE + 1];
  size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
  size_t width_stride = gridDim.x * blockDim.x;
  size_t full_width = (width & (~((unsigned long long)(BLOCK_SIZE - 1)))) + ((width & (BLOCK_SIZE - 1)) ? BLOCK_SIZE : 0); // round up to next block
  for (size_t w = idx; w < full_width; w += width_stride)
  { // grid-stride loop across matrix width
    sdata[threadIdx.y][threadIdx.x] = 0;
    size_t in_ptr = w + threadIdx.y * width;
    for (size_t h = threadIdx.y; h < height; h += BLOCK_SIZE)
    { // block-stride loop across matrix height
      sdata[threadIdx.y][threadIdx.x] += (w < width) ? in[in_ptr] : 0;
      in_ptr += width * BLOCK_SIZE;
    }
    __syncthreads();
    T my_val = sdata[threadIdx.x][threadIdx.y];
    for (int i = warpSize >> 1; i > 0; i >>= 1) // warp-wise parallel sum reduction
      my_val += __shfl_xor_sync(0xFFFFFFFFU, my_val, i);
    __syncthreads();
    if (threadIdx.x == 0)
      sdata[0][threadIdx.y] = my_val;
    __syncthreads();
    if ((threadIdx.y == 0) && ((w) < width))
      out[w] = sdata[0][threadIdx.x];
  }
}

__global__ void gpu_matrix_mult(float *a, float *b, float *c, int m, int n, int k)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  float sum = 0;
  if (col < k && row < m)
  {
    for (int i = 0; i < n; i++)
    {
      sum += a[row * n + i] * b[i * k + col];
    }
    c[row * k + col] = sum;
  }
}

__global__ void gpu_diag_matrix_mult(float *Val, int *RowPtr, float *diagVals, int total_rows)
{
  const int row = threadIdx.x + blockIdx.x * blockDim.x;
  if (row < total_rows)
  {
    for (int i = RowPtr[row]; i < RowPtr[row + 1]; i++)
    {
      Val[i] = diagVals[row] * Val[i];
    }
  }
}

///////////////////////////////////////////////////////////////////////////
// Sonar Claculation Function
__global__ void sonar_calculation(thrust::complex<float> *P_Beams,
                                  float *depth_image,
                                  float *normal_image,
                                  int width,
                                  int height,
                                  int depth_image_step,
                                  int normal_image_step,
                                  float *rand_image,
                                  int rand_image_step,
                                  float *reflectivity_image,
                                  int reflectivity_image_step,
                                  float hPixelSize,
                                  float vPixelSize,
                                  float hFOV,
                                  float vFOV,
                                  float beam_azimuthAngleWidth,
                                  float beam_elevationAngleWidth,
                                  float ray_azimuthAngleWidth,
                                  float ray_elevationAngleWidth,
                                  float soundSpeed,
                                  float sourceTerm,
                                  int nBeams, int nRays,
                                  int raySkips,
                                  float sonarFreq, float delta_f,
                                  int nFreq, float bandwidth,
                                  float attenuation,
                                  float area_scaler)
{
  // 2D Index of current thread
  const int beam = blockIdx.x * blockDim.x + threadIdx.x;
  const int ray = blockIdx.y * blockDim.y + threadIdx.y;

  //Only valid threads perform memory I/O
  if ((beam < width) && (ray < height) && (ray % raySkips == 0))
  {
    // Location of the image pixel
    const int depth_index = ray * depth_image_step / sizeof(float) + beam;
    const int normal_index = ray * normal_image_step / sizeof(float) + (3 * beam);
    const int rand_index = ray * rand_image_step / sizeof(float) + (2 * beam);
    const int reflectivity_index = ray * reflectivity_image_step / sizeof(float) + beam;

    // Input parameters for ray processing
    float distance = depth_image[depth_index] * 1.0f;
    float normal[3] = {normal_image[normal_index],
                      normal_image[normal_index + 1],
                      normal_image[normal_index + 2]};

    // Calculate ray angles
    double fl = static_cast<double>(width) / (2.0 * tan(hFOV/2.0));
    float ray_azimuthAngle = atan2(static_cast<double>(beam) -
                        0.5 * static_cast<double>(width-1), fl);
    float ray_elevationAngle = atan2(static_cast<double>(ray) -
                        0.5 * static_cast<double>(height-1), fl);

    // Beam pattern
    // float azimuthBeamPattern = abs(unnormalized_sinc(M_PI * 0.884
    // 				/ ray_azimuthAngleWidth * sin(ray_azimuthAngle)));
    // only one column of rays for each beam at beam center, interference calculated later
    float azimuthBeamPattern = 1.0;
    float elevationBeamPattern = abs(unnormalized_sinc(M_PI * 0.884
      				                    / (beam_elevationAngleWidth) * sin(ray_elevationAngle)));

    // incidence angle
    float incidence = acos(normal[2]); // compute_incidence(ray_azimuthAngle, ray_elevationAngle, normal);

    // ----- Point scattering model ------ //
    // Gaussian noise generated using opencv RNG
    float xi_z = rand_image[rand_index];
    float xi_y = rand_image[rand_index + 1];

    // Calculate amplitude
    thrust::complex<float> randomAmps = thrust::complex<float>(xi_z / sqrt(2.0), xi_y / sqrt(2.0));
    thrust::complex<float> lambert_sqrt =
        thrust::complex<float>(sqrt(reflectivity_image[reflectivity_index]) * cos(incidence), 0.0);
    thrust::complex<float> beamPattern =
        thrust::complex<float>(azimuthBeamPattern * elevationBeamPattern, 0.0);
    thrust::complex<float> targetArea_sqrt = thrust::complex<float>(sqrt(distance * area_scaler), 0.0);
    thrust::complex<float> propagationTerm =
        thrust::complex<float>(1.0 / pow(distance, 2.0) * exp(-2.0 * attenuation * distance), 0.0);
    thrust::complex<float> amplitude = randomAmps * thrust::complex<float>(sourceTerm, 0.0)
                                     * propagationTerm * beamPattern * lambert_sqrt * targetArea_sqrt;

    // Summation of Echo returned from a signal (frequency domain)
    for (size_t f = 0; f < nFreq; f++)
    {
      float freq;
      if (nFreq % 2 == 0)
        freq = delta_f * (-nFreq / 2.0 + f*1.0f + 1.0);
      else
        freq = delta_f * (-(nFreq - 1) / 2.0 + f*1.0f + 1.0);
      float kw = 2.0 * M_PI * freq / soundSpeed; // wave vector

      // Transmit spectrum, frequency domain
      thrust::complex<float> kernel = exp(thrust::complex<float>(0.0f, 2.0f * distance * kw)) * amplitude;
      P_Beams[beam * nFreq * (int)(nRays / raySkips) + (int)(ray / raySkips) * nFreq + f] =
          thrust::complex<float>(kernel.real() , kernel.imag());
    }
  }
}

///////////////////////////////////////////////////////////////////////////
namespace NpsGazeboSonar
{

  // CUDA Device Checker Wrapper
  void check_cuda_init_wrapper(void)
  {
    // Check CUDA device
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
      fprintf(stderr, "ERROR: %s\n", hipGetErrorString(error));
      exit(-1);
    }
  }

  // Sonar Claculation Function Wrapper
  CArray2D sonar_calculation_wrapper(const cv::Mat &depth_image,
                                     const cv::Mat &normal_image,
                                     const cv::Mat &rand_image,
                                     double _hPixelSize,
                                     double _vPixelSize,
                                     double _hFOV,
                                     double _vFOV,
                                     double _beam_azimuthAngleWidth,
                                     double _beam_elevationAngleWidth,
                                     double _ray_azimuthAngleWidth,
                                     double _ray_elevationAngleWidth,
                                     double _soundSpeed,
                                     double _maxDistance,
                                     double _sourceLevel,
                                     int _nBeams, int _nRays,
                                     int _raySkips,
                                     double _sonarFreq,
                                     double _bandwidth,
                                     int _nFreq,
                                     const cv::Mat &reflectivity_image,
                                     double _attenuation,
                                     float *window,
                                     float **beamCorrector,
                                     float beamCorrectorSum,
                                     bool debugFlag)
  {
    auto start = std::chrono::high_resolution_clock::now();
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    if (debugFlag)
      start = std::chrono::high_resolution_clock::now();

    // ----  Allocation of properties parameters  ---- //
    const float hPixelSize = (float)_hPixelSize;
    const float vPixelSize = (float)_vPixelSize;
    const float hFOV = (float)_hFOV;
    const float vFOV = (float)_vFOV;
    const float beam_elevationAngleWidth = (float)_beam_elevationAngleWidth;
    const float beam_azimuthAngleWidth = (float)_beam_azimuthAngleWidth;
    const float ray_elevationAngleWidth = (float)_ray_elevationAngleWidth;
    const float ray_azimuthAngleWidth = (float)_ray_azimuthAngleWidth;
    const float soundSpeed = (float)_soundSpeed;
    const float maxDistance = (float)_maxDistance;
    const float sonarFreq = (float)_sonarFreq;
    const float bandwidth = (float)_bandwidth;
    const float attenuation = (float)_attenuation;
    const int nBeams = _nBeams;
    const int nRays = _nRays;
    const int nFreq = _nFreq;
    const int raySkips = _raySkips;

    //#######################################################//
    //###############    Sonar Calculation   ################//
    //#######################################################//
    // ---------   Calculation parameters   --------- //
    const float max_distance = maxDistance;
    // Signal
    const float max_T = max_distance * 2.0 / soundSpeed;
    const float delta_f = 1.0 / max_T;
    // Precalculation
    const float area_scaler = ray_azimuthAngleWidth * ray_elevationAngleWidth;
    const float sourceLevel = (float)_sourceLevel;                     // db re 1 muPa;
    const float pref = 1e-6;                                           // 1 micro pascal (muPa);
    const float sourceTerm = sqrt(pow(10, (sourceLevel / 10))) * pref; // source term

    // ---------   Allocate GPU memory for image   --------- //
    //Calculate total number of bytes of input and output image
    const int depth_image_Bytes = depth_image.step * depth_image.rows;
    const int normal_image_Bytes = normal_image.step * normal_image.rows;
    const int rand_image_Bytes = rand_image.step * rand_image.rows;
    const int reflectivity_image_Bytes = reflectivity_image.step * reflectivity_image.rows;

    //Allocate device memory
    float *d_depth_image, *d_normal_image, *d_rand_image, *d_reflectivity_image;
    SAFE_CALL(hipMalloc((void **)&d_depth_image, depth_image_Bytes), "CUDA Malloc Failed");
    SAFE_CALL(hipMalloc((void **)&d_normal_image, normal_image_Bytes), "CUDA Malloc Failed");
    SAFE_CALL(hipMalloc((void **)&d_rand_image, rand_image_Bytes), "CUDA Malloc Failed");
    SAFE_CALL(hipMalloc((void **)&d_reflectivity_image, reflectivity_image_Bytes), "CUDA Malloc Failed");

    //Copy data from OpenCV input image to device memory
    SAFE_CALL(hipMemcpy(d_depth_image, depth_image.ptr(), depth_image_Bytes,
                  hipMemcpyHostToDevice), "CUDA Memcpy Failed");
    SAFE_CALL(hipMemcpy(d_normal_image, normal_image.ptr(), normal_image_Bytes,
                  hipMemcpyHostToDevice),"CUDA Memcpy Failed");
    SAFE_CALL(hipMemcpy(d_rand_image, rand_image.ptr(), rand_image_Bytes,
                  hipMemcpyHostToDevice),"CUDA Memcpy Failed");
    SAFE_CALL(hipMemcpy(d_reflectivity_image, reflectivity_image.ptr(), reflectivity_image_Bytes,
                  hipMemcpyHostToDevice), "CUDA Memcpy Failed");

    //Specify a reasonable block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    //Calculate grid size to cover the whole image
    const dim3 grid((depth_image.cols + block.x - 1) / block.x,
                    (depth_image.rows + block.y - 1) / block.y);

    // Beam data array
    thrust::complex<float> *P_Beams;
    thrust::complex<float> *d_P_Beams;
    const int P_Beams_N = nBeams * (int)(nRays / raySkips) * (nFreq + 1);
    const int P_Beams_Bytes = sizeof(thrust::complex<float>) * P_Beams_N;
    SAFE_CALL(hipHostMalloc((void **)&P_Beams, P_Beams_Bytes), "CUDA Malloc Failed");
    SAFE_CALL(hipMalloc((void **)&d_P_Beams, P_Beams_Bytes), "CUDA Malloc Failed");

    //Launch the beamor conversion kernel
    sonar_calculation<<<grid, block>>>(d_P_Beams,
                                       d_depth_image,
                                       d_normal_image,
                                       normal_image.cols,
                                       normal_image.rows,
                                       depth_image.step,
                                       normal_image.step,
                                       d_rand_image,
                                       rand_image.step,
                                       d_reflectivity_image,
                                       reflectivity_image.step,
                                       hPixelSize,
                                       vPixelSize,
                                       hFOV,
                                       vFOV,
                                       beam_azimuthAngleWidth,
                                       beam_elevationAngleWidth,
                                       ray_azimuthAngleWidth,
                                       ray_elevationAngleWidth,
                                       soundSpeed,
                                       sourceTerm,
                                       nBeams, nRays,
                                       raySkips,
                                       sonarFreq, delta_f,
                                       nFreq, bandwidth,
                                       attenuation,
                                       area_scaler);

    //Synchronize to check for any kernel launch errors
    SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");

    //Copy back data from destination device meory to OpenCV output image
    SAFE_CALL(hipMemcpy(P_Beams, d_P_Beams, P_Beams_Bytes,
                         hipMemcpyDeviceToHost), "CUDA Memcpy Failed");

    // Free GPU memory
    hipFree(d_depth_image);
    hipFree(d_normal_image);
    hipFree(d_rand_image);
    hipFree(d_reflectivity_image);
    hipFree(d_P_Beams);

    // For calc time measure
    if (debugFlag)
    {
      stop = std::chrono::high_resolution_clock::now();
      duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
      printf("GPU Sonar Computation Time %lld/100 [s]\n",
              static_cast<long long int>(duration.count() / 10000));
      start = std::chrono::high_resolution_clock::now();
    }

    //########################################################//
    //#########   Summation, Culling and windowing   #########//
    //########################################################//
    // Preallocate an array for return
    CArray2D P_Beams_F(CArray(nFreq), nBeams);
    // GPU grids and rows
    unsigned int grid_rows, grid_cols;
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    // GPU Ray summation using column sum
    float *P_Ray_real, *P_Ray_imag;
    float *d_P_Ray_real, *d_P_Ray_imag;
    const int P_Ray_N = (int)(nRays / raySkips) * (nFreq);
    const int P_Ray_Bytes = sizeof(float) * P_Ray_N;
    float *P_Ray_F_real, *P_Ray_F_imag;
    float *d_P_Ray_F_real, *d_P_Ray_F_imag;
    const int P_Ray_F_N = (nFreq)*1;
    const int P_Ray_F_Bytes = sizeof(float) * P_Ray_F_N;
    hipHostMalloc((void **)&P_Ray_real, P_Ray_Bytes);
    hipHostMalloc((void **)&P_Ray_imag, P_Ray_Bytes);
    hipHostMalloc((void **)&P_Ray_F_real, P_Ray_F_Bytes);
    hipHostMalloc((void **)&P_Ray_F_imag, P_Ray_F_Bytes);
    SAFE_CALL(hipMalloc((void **)&d_P_Ray_real, P_Ray_Bytes), "CUDA Malloc Failed");
    SAFE_CALL(hipMalloc((void **)&d_P_Ray_imag, P_Ray_Bytes), "CUDA Malloc Failed");
    SAFE_CALL(hipMalloc((void **)&d_P_Ray_F_real, P_Ray_F_Bytes), "CUDA Malloc Failed");
    SAFE_CALL(hipMalloc((void **)&d_P_Ray_F_imag, P_Ray_F_Bytes), "CUDA Malloc Failed");

    dim3 dimGrid_Ray((nFreq + BLOCK_SIZE - 1) / BLOCK_SIZE);

    for (size_t beam = 0; beam < nBeams; beam ++)
    {
      for (size_t ray = 0; ray < (int)(nRays / raySkips); ray++)
      {
        for (size_t f = 0; f < nFreq; f++)
        {
          P_Ray_real[ray * nFreq + f] =
              P_Beams[beam * nFreq * (int)(nRays / raySkips) + ray * nFreq + f].real();
          P_Ray_imag[ray * nFreq + f] =
              P_Beams[beam * nFreq * (int)(nRays / raySkips) + ray * nFreq + f].imag();
        }
      }

      SAFE_CALL(hipMemcpy(d_P_Ray_real, P_Ray_real, P_Ray_Bytes, hipMemcpyHostToDevice),
                "CUDA Memcpy Failed");
      SAFE_CALL(hipMemcpy(d_P_Ray_imag, P_Ray_imag, P_Ray_Bytes, hipMemcpyHostToDevice),
                "CUDA Memcpy Failed");

      column_sums_reduce<<<dimGrid_Ray, dimBlock>>>(d_P_Ray_real, d_P_Ray_F_real,
                                                    nFreq, (int)(nRays / raySkips));
      column_sums_reduce<<<dimGrid_Ray, dimBlock>>>(d_P_Ray_imag, d_P_Ray_F_imag,
                                                    nFreq, (int)(nRays / raySkips));

      SAFE_CALL(hipMemcpy(P_Ray_F_real, d_P_Ray_F_real, P_Ray_F_Bytes,
                           hipMemcpyDeviceToHost), "CUDA Memcpy Failed");
      SAFE_CALL(hipMemcpy(P_Ray_F_imag, d_P_Ray_F_imag, P_Ray_F_Bytes,
                           hipMemcpyDeviceToHost), "CUDA Memcpy Failed");
      SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");

      for (size_t f = 0; f < nFreq; f++)
        P_Beams_F[beam][f] = Complex(P_Ray_F_real[f], P_Ray_F_imag[f]);
    }

    // free memory
    hipHostFree(P_Beams);
    hipHostFree(P_Ray_real);
    hipHostFree(P_Ray_imag);
    hipHostFree(P_Ray_F_real);
    hipHostFree(P_Ray_F_imag);
    hipFree(d_P_Ray_real);
    hipFree(d_P_Ray_imag);
    hipFree(d_P_Ray_F_real);
    hipFree(d_P_Ray_F_imag);

    if (debugFlag)
    {
      stop = std::chrono::high_resolution_clock::now();
      duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
      printf("Sonar Ray Summation %lld/100 [s]\n",
            static_cast<long long int>(duration.count() / 10000));
      start = std::chrono::high_resolution_clock::now();
    }

    // -------------- Beam culling correction -----------------//
    // beamCorrector and beamCorrectorSum is precalculated at parent cpp
    float *P_Beams_Cor_real, *P_Beams_Cor_imag;
    float *P_Beams_Cor_F_real, *P_Beams_Cor_F_imag;
    float *P_Beams_Cor_real_tmp, *P_Beams_Cor_imag_tmp;
    float *d_P_Beams_Cor_real, *d_P_Beams_Cor_imag;
    float *d_P_Beams_Cor_F_real, *d_P_Beams_Cor_F_imag;
    const int P_Beams_Cor_N = nBeams * nFreq;
    const int P_Beams_Cor_Bytes = sizeof(float) * P_Beams_Cor_N;
    hipHostMalloc((void **)&P_Beams_Cor_real, P_Beams_Cor_Bytes);
    hipHostMalloc((void **)&P_Beams_Cor_imag, P_Beams_Cor_Bytes);
    hipHostMalloc((void **)&P_Beams_Cor_real_tmp, P_Beams_Cor_Bytes);
    hipHostMalloc((void **)&P_Beams_Cor_imag_tmp, P_Beams_Cor_Bytes);
    hipHostMalloc((void **)&P_Beams_Cor_F_real, P_Beams_Cor_Bytes);
    hipHostMalloc((void **)&P_Beams_Cor_F_imag, P_Beams_Cor_Bytes);
    SAFE_CALL(hipMalloc((void **)&d_P_Beams_Cor_real, P_Beams_Cor_Bytes), "CUDA Malloc Failed");
    SAFE_CALL(hipMalloc((void **)&d_P_Beams_Cor_imag, P_Beams_Cor_Bytes), "CUDA Malloc Failed");
    SAFE_CALL(hipMalloc((void **)&d_P_Beams_Cor_F_real, P_Beams_Cor_Bytes), "CUDA Malloc Failed");
    SAFE_CALL(hipMalloc((void **)&d_P_Beams_Cor_F_imag, P_Beams_Cor_Bytes), "CUDA Malloc Failed");

    float *beamCorrector_lin, *d_beamCorrector_lin;
    const int beamCorrector_lin_N = nBeams * nBeams;
    const int beamCorrector_lin_Bytes = sizeof(float) * beamCorrector_lin_N;
    hipHostMalloc((void **)&beamCorrector_lin, beamCorrector_lin_Bytes);
    SAFE_CALL(hipMalloc((void **)&d_beamCorrector_lin, beamCorrector_lin_Bytes), "CUDA Malloc Failed");

    // (nfreq x nBeams) * (nBeams x nBeams) = (nfreq x nBeams)
    for (size_t beam = 0; beam < nBeams; beam ++)
    {
      for (size_t f = 0; f < nFreq; f++)
      {
        P_Beams_Cor_real[f * nBeams + beam] = P_Beams_F[beam][f].real() * 1.0f;
        P_Beams_Cor_imag[f * nBeams + beam] = P_Beams_F[beam][f].imag() * 1.0f;
      }
      for (size_t beam_other = 0; beam_other < nBeams; beam_other ++)
        beamCorrector_lin[beam_other * nBeams + beam] = beamCorrector[beam][beam_other];
    }

    SAFE_CALL(hipMemcpy(d_P_Beams_Cor_real, P_Beams_Cor_real, P_Beams_Cor_Bytes,
                         hipMemcpyHostToDevice),
              "CUDA Memcpy Failed");
    SAFE_CALL(hipMemcpy(d_P_Beams_Cor_imag, P_Beams_Cor_imag, P_Beams_Cor_Bytes,
                         hipMemcpyHostToDevice),
              "CUDA Memcpy Failed");
    SAFE_CALL(hipMemcpy(d_beamCorrector_lin, beamCorrector_lin, beamCorrector_lin_Bytes,
                         hipMemcpyHostToDevice),
              "CUDA Memcpy Failed");

    grid_rows = (nFreq + BLOCK_SIZE - 1) / BLOCK_SIZE;
    grid_cols = (nBeams + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid_Beam(grid_cols, grid_rows);

    gpu_matrix_mult<<<dimGrid_Beam, dimBlock>>>(d_P_Beams_Cor_real, d_beamCorrector_lin,
                                                d_P_Beams_Cor_F_real, nFreq, nBeams, nBeams);
    SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");

    gpu_matrix_mult<<<dimGrid_Beam, dimBlock>>>(d_P_Beams_Cor_imag, d_beamCorrector_lin,
                                                d_P_Beams_Cor_F_imag, nFreq, nBeams, nBeams);
    SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");

    //Copy back data from destination device meory
    SAFE_CALL(hipMemcpy(P_Beams_Cor_real_tmp, d_P_Beams_Cor_F_real, P_Beams_Cor_Bytes,
                         hipMemcpyDeviceToHost),
              "CUDA Memcpy Failed");
    SAFE_CALL(hipMemcpy(P_Beams_Cor_imag_tmp, d_P_Beams_Cor_F_imag, P_Beams_Cor_Bytes,
                         hipMemcpyDeviceToHost),
              "CUDA Memcpy Failed");
    SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");

    // ---------------    Windowing   ----------------- //
    float *window_diag, *d_window;
    const int window_N = nFreq * 1;
    const int window_Bytes = sizeof(float) * window_N;
    window_diag = (float *)malloc(window_Bytes);
    SAFE_CALL(hipMalloc((void **)&d_window, window_Bytes), "CUDA Malloc Failed");

    int *diag_ptr, *d_diag_ptr;
    const int diag_ptr_N = nBeams * 1;
    const int diag_ptr_Bytes = sizeof(int) * diag_ptr_N;
    diag_ptr = (int *)malloc(diag_ptr_Bytes);
    SAFE_CALL(hipMalloc((void **)&d_diag_ptr, diag_ptr_Bytes), "CUDA Malloc Failed");

    // (nBeams x nfreq) * (1 x nFreq) = (nBeams x nFreq)
    for (size_t beam = 0; beam < nBeams; beam ++)
    {
      for (size_t f = 0; f < nFreq; f++)
      { // Transpose
        P_Beams_Cor_real[beam * nFreq + f] = P_Beams_Cor_real_tmp[f * nBeams + beam];
        P_Beams_Cor_imag[beam * nFreq + f] = P_Beams_Cor_imag_tmp[f * nBeams + beam];
        window_diag[f] = window[f];
      }
      diag_ptr[beam] = (int)beam;
    }
    SAFE_CALL(hipMemcpy(d_P_Beams_Cor_real, P_Beams_Cor_real, P_Beams_Cor_Bytes,
                         hipMemcpyHostToDevice),
              "CUDA Memcpy Failed");
    SAFE_CALL(hipMemcpy(d_P_Beams_Cor_imag, P_Beams_Cor_imag, P_Beams_Cor_Bytes,
                         hipMemcpyHostToDevice),
              "CUDA Memcpy Failed");
    SAFE_CALL(hipMemcpy(d_window, window_diag, window_Bytes,
                         hipMemcpyHostToDevice),
              "CUDA Memcpy Failed");
    SAFE_CALL(hipMemcpy(d_diag_ptr, diag_ptr, diag_ptr_Bytes,
                         hipMemcpyHostToDevice),
              "CUDA Memcpy Failed");

    grid_rows = (nFreq + BLOCK_SIZE - 1) / BLOCK_SIZE;
    grid_cols = (nFreq + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid_window(grid_cols, grid_rows);
    gpu_diag_matrix_mult<<<dimGrid_window, dimBlock>>>(d_P_Beams_Cor_real, d_diag_ptr, d_window, nFreq);
    SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");

    gpu_diag_matrix_mult<<<dimGrid_window, dimBlock>>>(d_P_Beams_Cor_imag, d_diag_ptr, d_window, nFreq);
    SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");

    //Copy back data from destination device meory
    SAFE_CALL(hipMemcpy(P_Beams_Cor_F_real, d_P_Beams_Cor_real, P_Beams_Cor_Bytes,
                         hipMemcpyDeviceToHost),
              "CUDA Memcpy Failed");
    SAFE_CALL(hipMemcpy(P_Beams_Cor_F_imag, d_P_Beams_Cor_imag, P_Beams_Cor_Bytes,
                         hipMemcpyDeviceToHost),
              "CUDA Memcpy Failed");
    SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");

    // Return
    for (size_t beam = 0; beam < nBeams; beam ++)
      for (size_t f = 0; f < nFreq; f++)
        P_Beams_F[beam][f] =
            Complex(P_Beams_Cor_F_real[beam * nFreq + f] / beamCorrectorSum,
              P_Beams_Cor_F_imag[beam * nFreq + f] / beamCorrectorSum);

    // Free memory
    hipFree(d_P_Beams_Cor_imag);
    hipFree(d_P_Beams_Cor_real);
    hipFree(d_P_Beams_Cor_F_imag);
    hipFree(d_P_Beams_Cor_F_real);
    hipFree(d_beamCorrector_lin);
    hipFree(d_window);
    hipFree(d_diag_ptr);
    hipHostFree(P_Beams_Cor_real);
    hipHostFree(P_Beams_Cor_imag);
    hipHostFree(P_Beams_Cor_F_real);
    hipHostFree(P_Beams_Cor_F_imag);
    hipHostFree(P_Beams_Cor_real_tmp);
    hipHostFree(P_Beams_Cor_imag_tmp);
    hipHostFree(beamCorrector_lin);
    hipHostFree(window_diag);
    hipHostFree(diag_ptr);

    // For calc time measure
    if (debugFlag)
    {
      stop = std::chrono::high_resolution_clock::now();
      duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
      printf("GPU Window & Correction %lld/100 [s]\n",
            static_cast<long long int>(duration.count() / 10000));
      start = std::chrono::high_resolution_clock::now();
    }

    //#################################################//
    //###################   FFT   #####################//
    //#################################################//
    SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");
    const int DATASIZE = nFreq;
    const int BATCH = nBeams;
    // --- Host side input data allocation and initialization
    hipfftComplex *hostInputData = (hipfftComplex *)malloc(
        DATASIZE * BATCH * sizeof(hipfftComplex));
    for (int beam = 0; beam < BATCH; beam++)
    {
      for (int f = 0; f < DATASIZE; f++)
      {
        if (f < nFreq)
          hostInputData[beam * DATASIZE + f] =
              make_hipComplex(P_Beams_F[beam][f].real() * 1.0f,
                             P_Beams_F[beam][f].imag() * 1.0f);
        else
          hostInputData[beam * DATASIZE + f] =
              (make_hipComplex(0.f, 0.f)); // zero padding
      }
    }

    // --- Device side input data allocation and initialization
    hipfftComplex *deviceInputData;
    SAFE_CALL(hipMalloc((void **)&deviceInputData,
                         DATASIZE * BATCH * sizeof(hipfftComplex)),
                         "FFT CUDA Malloc Failed");
    SAFE_CALL(hipMemcpy(deviceInputData, hostInputData,
                         DATASIZE * BATCH * sizeof(hipfftComplex),
                         hipMemcpyHostToDevice),
                         "FFT CUDA Memcopy Failed");

    // --- Host side output data allocation
    hipfftComplex *hostOutputData =
        (hipfftComplex *)malloc(DATASIZE * BATCH * sizeof(hipfftComplex));

    // --- Device side output data allocation
    hipfftComplex *deviceOutputData;
    hipMalloc((void **)&deviceOutputData,
               DATASIZE * BATCH * sizeof(hipfftComplex));

    // --- Batched 1D FFTs
    hipfftHandle handle;
    int rank = 1;         // --- 1D FFTs
    int n[] = {DATASIZE}; // --- Size of the Fourier transform
    // --- Distance between two successive input/output elements
    int istride = 1, ostride = 1;
    int idist = DATASIZE, odist = DATASIZE; // --- Distance between batches
    // --- Input/Output size with pitch (ignored for 1D transforms)
    int inembed[] = {0};
    int onembed[] = {0};
    int batch = BATCH; // --- Number of batched executions
    hipfftPlanMany(&handle, rank, n,
                  inembed, istride, idist,
                  onembed, ostride, odist, HIPFFT_C2C, batch);

    hipfftExecC2C(handle, deviceInputData, deviceOutputData, HIPFFT_FORWARD);

    // --- Device->Host copy of the results
    SAFE_CALL(hipMemcpy(hostOutputData, deviceOutputData,
                         DATASIZE * BATCH * sizeof(hipfftComplex),
                         hipMemcpyDeviceToHost),
                         "FFT CUDA Memcopy Failed");

    hipfftDestroy(handle);
    hipFree(deviceOutputData);
    hipFree(deviceInputData);
    free(hostInputData);
    free(hostOutputData);


    for (int beam = 0; beam < BATCH; beam++)
    {
      for (int f = 0; f < nFreq; f++)
      {
        P_Beams_F[beam][f] =
            Complex(hostOutputData[beam * DATASIZE + f].x * delta_f,
                    hostOutputData[beam * DATASIZE + f].y * delta_f);
      }
    }

    // For calc time measure
    if (debugFlag)
    {
      stop = std::chrono::high_resolution_clock::now();
      duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
      printf("GPU FFT Calc Time %lld/100 [s]\n",
            static_cast<long long int>(duration.count() / 10000));
    }

    return P_Beams_F;
  }
} // namespace NpsGazeboSonar
